#include "cublas_sgemm.h"
#include "hipblas.h"

int cublas_sgemm(const CUBLAS_LAYOUT Layout, const CUBLAS_TRANSPOSE TransA,
                 const CUBLAS_TRANSPOSE TransB, const int M, const int N,
                 const int K, const float alpha, const float *A,
                 const int lda, const float *B, const int ldb,
                 const float beta, float *C, const int ldc)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    float *devPtrA, *devPtrB, *devPtrC;
    cudaStat = hipMalloc((void **)&devPtrA, M * K * sizeof(*A));
    if (cudaStat != hipSuccess)
    {
        return -cudaStat;
    }
    cudaStat = hipMalloc((void **)&devPtrB, K * N * sizeof(*B));
    if (cudaStat != hipSuccess)
    {
        hipFree(devPtrA);
        return -cudaStat;
    }
    cudaStat = hipMalloc((void **)&devPtrC, M * N * sizeof(*C));
    if (cudaStat != hipSuccess)
    {
        hipFree(devPtrA);
        hipFree(devPtrB);
        return -cudaStat;
    }

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        return stat;
    }
    if (TransA == CublasNoTrans)
    {
        stat = hipblasSetMatrix(K, M, sizeof(*A), A, K, devPtrA, K);
    }
    else if (TransA == CublasTrans)
    {
        stat = hipblasSetMatrix(M, K, sizeof(*A), A, M, devPtrA, M);
    }
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        hipblasDestroy(handle);
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        return stat;
    }
    if (TransB == CublasNoTrans)
    {
        stat = hipblasSetMatrix(N, K, sizeof(*B), B, N, devPtrB, N);
    }
    else if (TransB == CublasTrans)
    {
        stat = hipblasSetMatrix(K, N, sizeof(*B), B, K, devPtrB, K);
    }
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        hipblasDestroy(handle);
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        return stat;
    }

    if (TransA == CublasNoTrans && TransB == CublasNoTrans)
    {
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, devPtrB, N, devPtrA, K, &beta, devPtrC, N);
    }
    else if (TransA == CublasNoTrans && TransB == CublasTrans)
    {
        stat = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, K, &alpha, devPtrB, K, devPtrA, K, &beta, devPtrC, N);
    }
    else if (TransA == CublasTrans && TransB == CublasNoTrans)
    {
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, M, K, &alpha, devPtrB, N, devPtrA, M, &beta, devPtrC, N);
    }
    else if (TransA == CublasTrans && TransB == CublasTrans)
    {
        stat = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, M, K, &alpha, devPtrB, K, devPtrA, M, &beta, devPtrC, N);
    }
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        hipblasDestroy(handle);
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        return stat;
    }

    stat = hipblasGetMatrix(M, N, sizeof(*C), devPtrC, M, C, M);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        hipblasDestroy(handle);
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        return stat;
    }

    hipblasDestroy(handle);
    hipFree(devPtrA);
    hipFree(devPtrB);
    hipFree(devPtrC);
    return HIPBLAS_STATUS_SUCCESS;
}
